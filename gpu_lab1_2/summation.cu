#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdlib.h>

struct results
{
	float sum;
};

#include "summation_kernel.cu"

// CPU implementation
float log2_series(int n)
{
	int i;
	float sum = 0;
	for (i = 0; i < n; i++) {
		if (i % 2 == 0)
			sum += 1.0 / (i + 1);
		else
			sum -= 1.0 / (i + 1);
	}
	return sum;
}

float reverse_log2_series(int n)
{
	int i;
	float sum = 0;
	for (i = n-1; i >= 0; i--) {
		if (i % 2 == 0)
			sum += 1.0 / (i + 1);
		else
			sum -= 1.0 / (i + 1);
	}
	return sum;
}

void basic_gpu_summation(int data_size)
{
	int i;
    // Parameter definition
    int threads_per_block = 16 * 32;
    int blocks_in_grid = 8;
    
    int num_threads = threads_per_block * blocks_in_grid;

    // Timer initialization
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));

	// Allocating output data on GPU
    float *d_C;
	hipMalloc((void **)&d_C, num_threads * sizeof(float));

    // Start timer
    CUDA_SAFE_CALL(hipEventRecord(start, 0));

    // Execute kernel
    summation_kernel<<<blocks_in_grid,threads_per_block>>>(data_size, d_C);

    // Stop timer
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));

    // Get results back
    float* res = (float*)malloc(num_threads * sizeof(float));
	hipMemcpy(res, d_C, num_threads * sizeof(float), hipMemcpyDeviceToHost);
	
    // Finish reduction
	float sum = 0.;
	for(i = 0; i < num_threads; i ++) {
		sum += res[i];
	}
    
    // Cleanup
	hipFree(d_C);
	free(res);
    
    printf("GPU results:\n");
    printf(" Sum: %.15f\n", sum);
    
    float elapsedTime;
    CUDA_SAFE_CALL(hipEventElapsedTime(&elapsedTime, start, stop));	// In ms

    double total_time = elapsedTime / 1000.;	// s
    double time_per_iter = total_time / (double)data_size;
    double bandwidth = sizeof(float) / time_per_iter; // B/s
    
    printf(" Total time: %g s,\n Per iteration: %g ns\n Throughput: %g GB/s\n",
    	total_time,
    	time_per_iter * 1.e9,
    	bandwidth / 1.e9);
  
    CUDA_SAFE_CALL(hipEventDestroy(start));
    CUDA_SAFE_CALL(hipEventDestroy(stop));
}

void reduced_gpu_summation(int data_size)
{
	int i;
    // Parameter definition
    int threads_per_block = 16 * 32;
    int blocks_in_grid = 8;

    // Timer initialization
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));

	// Allocating output data on GPU
    float *d_C;
	hipMalloc((void **)&d_C, blocks_in_grid * sizeof(float));

	// Shared memory size
	int smemSize = threads_per_block * sizeof(float);

    // Start timer
    CUDA_SAFE_CALL(hipEventRecord(start, 0));

    // Execute kernel
    reduced_summation_kernel<<<blocks_in_grid, threads_per_block, smemSize>>>(data_size, d_C);

    // Stop timer
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));

    // Get results back
    float* res = (float*)malloc(blocks_in_grid * sizeof(float));
	hipMemcpy(res, d_C, blocks_in_grid * sizeof(float), hipMemcpyDeviceToHost);
	
    // Finish reduction
	float sum = 0.;
	for(i = 0; i < blocks_in_grid; i ++) {
		sum += res[i];
	}
    
    // Cleanup
	hipFree(d_C);
	free(res);
    
    printf("Reduced GPU results:\n");
    printf(" Sum: %.15f\n", sum);
    
    float elapsedTime;
    CUDA_SAFE_CALL(hipEventElapsedTime(&elapsedTime, start, stop));	// In ms

    double total_time = elapsedTime / 1000.;	// s
    double time_per_iter = total_time / (double)data_size;
    double bandwidth = sizeof(float) / time_per_iter; // B/s
    
    printf(" Total time: %g s,\n Per iteration: %g ns\n Throughput: %g GB/s\n",
    	total_time,
    	time_per_iter * 1.e9,
    	bandwidth / 1.e9);
  
    CUDA_SAFE_CALL(hipEventDestroy(start));
    CUDA_SAFE_CALL(hipEventDestroy(stop));
}

void full_gpu_summation(int data_size)
{
	int i;
    // Parameter definition
    int threads_per_block = 16 * 32;
    int blocks_in_grid = 8;

    // Timer initialization
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));

	// Allocating output data on GPU
    float *d_C;
	hipMalloc((void **)&d_C, blocks_in_grid * sizeof(float));

	// Shared memory size
	int smemSize = threads_per_block * sizeof(float);

    // Start timer
    CUDA_SAFE_CALL(hipEventRecord(start, 0));

    // Execute kernel
    reduced_summation_kernel<<<blocks_in_grid, threads_per_block, smemSize>>>(data_size, d_C);

    // Stop timer
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));

    // Get results back
	int resSize = blocks_in_grid * sizeof(float);
    float* res = (float*)malloc(resSize);
	hipMemcpy(res, d_C, resSize, hipMemcpyDeviceToHost);
	
	// On renvoie le tableau de r�sultats des threads au GPU (sur un seul block) pour faire la somme des r�sultats.
	float *d_res, *d_sum_result;
	hipMalloc((void**)&d_res, resSize);
	hipMalloc((void**)&d_sum_result, sizeof(float));

	hipMemcpy(d_res, res, resSize, hipMemcpyHostToDevice);

	smemSize = blocks_in_grid * sizeof(float);
    reduced_array_summation<<<1, blocks_in_grid, smemSize>>>(d_res, d_sum_result);

	float sum_result;
	hipMemcpy(&sum_result, d_sum_result, sizeof(float), hipMemcpyDeviceToHost);

    printf("full GPU results:\n");
    printf(" Sum: %.15f\n", sum_result);

    // Cleanup
	hipFree(d_C);
	hipFree(d_res);
	free(res);
    
    float elapsedTime;
    CUDA_SAFE_CALL(hipEventElapsedTime(&elapsedTime, start, stop));	// In ms

    double total_time = elapsedTime / 1000.;	// s
    double time_per_iter = total_time / (double)data_size;
    double bandwidth = sizeof(float) / time_per_iter; // B/s
    
    printf(" Total time: %g s,\n Per iteration: %g ns\n Throughput: %g GB/s\n",
    	total_time,
    	time_per_iter * 1.e9,
    	bandwidth / 1.e9);
  
    CUDA_SAFE_CALL(hipEventDestroy(start));
    CUDA_SAFE_CALL(hipEventDestroy(stop));
}

int main(int argc, char ** argv)
{
    int data_size = 1024 * 1024 * 128;

    // Run CPU version
    double start_time = getclock();
    float log2 = log2_series(data_size);
    double end_time = getclock();

    double r_start_time = getclock();
    float r_log2 = reverse_log2_series(data_size);
    double r_end_time = getclock();
    
    printf(" log(2)=\n%.15f\n", log(2.0));
    printf("CPU result:\n%.15f\n", log2);
    printf("CPU result (reverse):\n%.15f\n", r_log2);
    printf(" time=%fs\n", end_time - start_time);
    printf(" time=%fs (reverse)\n", r_end_time - r_start_time);
    
	reduced_gpu_summation(data_size);
	// full_gpu_summation(data_size);

    return 0;
}

