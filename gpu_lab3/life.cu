
#include "utils.h"
#include <stdlib.h>

#include "life_kernel.cu"


int main(int argc, char ** argv)
{
    // Definition of parameters
    int domain_x = 128;	// Multiple of threads_per_block * cells_per_word
    int domain_y = 128;
    
    int cells_per_word = 1;
    
    int steps = 2;
    
	int threads_x = 16;
	int threads_y = 16;
    int threads_per_block = threads_x * threads_y;
    int blocks_x = domain_x / (threads_x * cells_per_word);
    int blocks_y = domain_y / (threads_y * cells_per_word);
    
    dim3  grid(blocks_x, blocks_y);	// CUDA grid dimensions
	dim3  gridInit(1, domain_y);
    dim3  threads(threads_x, threads_y); // CUDA block dimensions
	dim3  threadsInit(128);

    // Allocation of arrays
    int * domain_gpu[2] = {NULL, NULL};

	// Arrays of dimensions domain.x * domain.y
	size_t domain_size = domain_x * domain_y / cells_per_word * sizeof(int);
	CUDA_SAFE_CALL(hipMalloc((void**)&domain_gpu[0], domain_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&domain_gpu[1], domain_size));

	init_kernel<<< gridInit, threadsInit, 0 >>>(domain_gpu[0], domain_x);

    // Timer initialization
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));

    // Start timer
    CUDA_SAFE_CALL(hipEventRecord(start, 0));

    // Kernel execution
    int shared_mem_size = (threads_x + 2) * (threads_y + 2) * sizeof(int); // cellules "actives" + cellules "bordures"
    for(int i = 0; i < steps; i++) {
	    life_kernel<<< grid, threads, shared_mem_size >>>(domain_gpu[i%2],
	    	domain_gpu[(i+1)%2], domain_x, domain_y);
	}

    // Stop timer
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));
    
    float elapsedTime;
    CUDA_SAFE_CALL(hipEventElapsedTime(&elapsedTime, start, stop));	// In ms
    printf("GPU time: %f ms\n", elapsedTime);

    CUDA_SAFE_CALL(hipEventDestroy(start));
    CUDA_SAFE_CALL(hipEventDestroy(stop));

    // Get results back
    int * domain_cpu = (int*)malloc(domain_size);
    CUDA_SAFE_CALL(hipMemcpy(domain_cpu, domain_gpu[steps%2], domain_size, hipMemcpyDeviceToHost));

    CUDA_SAFE_CALL(hipFree(domain_gpu[0]));
    CUDA_SAFE_CALL(hipFree(domain_gpu[1]));

    // Count colors
    int red = 0;
    int blue = 0;
    for(int y = 0; y < domain_y; y++)
    {
    	for(int x = 0; x < domain_x; x++)
    	{
    		int cell = domain_cpu[y * domain_x + x];
    		//printf("%u", cell);
    		if(cell == 1) {
    			red++;
    		}
    		else if(cell == 2) {
    			blue++;
    		}
    	}
    	//printf("\n");
    }

    printf("Red/Blue cells: %d/%d\n", red, blue);
    
    free(domain_cpu);
  
    return 0;
}

